
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#define MAX_NONCE 1000000000 // 100000000000

//char* tohexadecimal

void mine(long blockNum, char *trans, char *preHash, int prefixZero){
    //char prefix[] = "0000" ;
    
    for(int i = 0; i < MAX_NONCE; i++){
        //printf("mining...\n") ;
        srand(i*blockNum*(trans[0])*(preHash[0]));
        int count = 0 ;
        for(int j = 0; j < prefixZero; j++){
            if(rand() % 10 == 0){
                count++ ;
            }
        }
        if (count == prefixZero){
            //printf("found, nonce = %d\n", i) ;
            
        }
        //printf("%d\n", rand() % 10);
    }
}

int main(){
    char trans[] = "A-20->B,b-10->C" ;
    char preHash[] = "0000000xa036944e29568d0cff17edbe038f81208fecf9a66be9a2b8321c6ec7" ;

    int difficulty = 4 ;

    hipEvent_t start, stop ;
	hipEventCreate(&start) ;
	hipEventCreate(&stop) ;

    hipEventRecord(start) ;
    mine(1, trans, preHash, difficulty) ;
    hipEventRecord(stop) ;

    hipEventSynchronize(stop) ;
	float millisec = 0 ;
	hipEventElapsedTime(&millisec, start, stop) ;
    printf("Time used: %f\n", millisec) ;

    printf("end\n") ;

    return 0 ;
}
