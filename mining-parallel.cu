
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>
#define MAX_NONCE 100000000000 // 100000000000
#define MAX 10

//char* tohexadecimal

void mine1(long blockNum, char *trans, char *preHash, int prefixZero){
    //char prefix[] = "0000" ;
    
    for(int i = 0; i < MAX_NONCE; i++){
        printf("mining...\n") ;
        srand(i*blockNum*(trans[0])*(preHash[0]));
        int count = 0 ;
        for(int j = 0; j < prefixZero; j++){
            if(rand() % 10 == 0){
                count++ ;
            }
        }
        if (count == prefixZero){
            printf("found, nonce = %d\n", i) ;
            break;
        }
        //printf("%d\n", rand() % 10);
    }
}

__global__ void mine(long int* blockNum, char *trans, char *preHash, int *prefixZero){
    int index = threadIdx.x ;
    for(int i = 0; i < (MAX_NONCE/1024/10); i++){
        //printf("mining...\n") ;
        int n = ((MAX_NONCE/1024/10)*(blockIdx.x*blockDim.x)+index) + i ;
        hiprandState_t state;
        hiprand_init(n*(*blockNum)*(*trans)*(*preHash), 0, 0, &state);
        //printf("rand = %d\n", curand(&state) % MAX) ;
        //int random = curand(&state) % MAX ;
        //printf("random = %d\n", random) ;
        //srand(n*(*blockNum)*(*trans)*(*preHash));
        int count = 0 ;
        int random = hiprand(&state) % MAX ;
        for(int j = 0; j < (*prefixZero); j++){
            if(random == 0){
                count++ ;
            }
        }
        if (count == (*prefixZero)){
            //printf("found, nonce = %d\n", n) ;
            //exit(1) ;
        }
    }
}

int main(){
    char trans[] = "A-20->B,b-10->C" ;
    char preHash[] = "0000000xa036944e29568d0cff17edbe038f81208fecf9a66be9a2b8321c6ec7" ;

    int difficulty = 5 ;
    //mine(1, trans, preHash, difficulty) ;

    long int blockNum = 1 ;
    char tran = trans[0] ;
    char preH = preHash[0] ;

    long int *d_blockNum ;
    char *d_trans ;
    char *d_preHash ;
    int *d_diff ;

    hipMalloc((void**) &d_blockNum, sizeof(long int));
    hipMalloc((void**) &d_trans, sizeof(char));
    hipMalloc((void**) &d_preHash, sizeof(char));
    hipMalloc((void**) &d_diff, sizeof(int));
    hipMemcpy(d_blockNum, &blockNum, sizeof(long int), hipMemcpyHostToDevice);
    hipMemcpy(d_trans, &tran, sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_preHash, &preH, sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_diff, &difficulty, sizeof(int), hipMemcpyHostToDevice);

    hipEvent_t start, stop ;
	hipEventCreate(&start) ;
	hipEventCreate(&stop) ;

    hipEventRecord(start) ;
    mine<<<10, 1024>>>(d_blockNum, d_trans, d_preHash, d_diff) ;
    hipEventRecord(stop) ;

    hipEventSynchronize(stop) ;
	float millisec = 0 ;
	hipEventElapsedTime(&millisec, start, stop) ;
    printf("Time used: %f\n", millisec) ;

    hipFree(d_blockNum);
    hipFree(d_trans);
    hipFree(d_preHash);
    hipFree(d_diff);

    printf("end\n") ;

    return 0 ;
}